#include <algorithm>
#include "cuda_neuronalNetwork.h"
#include "cuda_kernel.h"

namespace cuda {

	neuronalNetwork::neuronalNetwork() {

	}

	void neuronalNetwork::set_config(const config_t config) {
		current_config_ = config;
	}

	void neuronalNetwork::train(cuda::model& model, std::vector<data::sample<float>>& trainings_data) {
		std::vector<float>& ref_input = trainings_data[0].internalData();

		model.init((ref_input.size() + 1) * current_config_.num_hidden
				+ (current_config_.num_hidden + 1) * current_config_.num_output);

		data_context context(current_config_, model, trainings_data);
		context.synchronize(current_config_, model, trainings_data);

		int i = 0;

		for (data::sample<float>& s : trainings_data) {
			train_sample(i, s, context, model);
			i++;
		}

		context.dev_weights.synchFromDevice(model.getWeights());
	}

	neuronalNetwork::test_result_t neuronalNetwork::test(cuda::model& model, std::vector<data::sample<float>>& test_data) {
		data_context context(current_config_, model, test_data);
		context.synchronize(current_config_, model, test_data);

		test_result_t result;
		int i = 0;

		for (data::sample<float>& s : test_data) {
			if (test_sample(i, s, context, model)) {
				result.correct++;
			}

			i++;
		}

		result.total = test_data.size();
		result.error = result.total - result.correct;
		result.ratio = (float) result.correct / (float) result.total;

		return result;
	}

	bool neuronalNetwork::train_sample(const int i, const data::sample<float>& sample, data_context& context, cuda::model& model) {
		int num_blocks;
		int num_threads;

		num_blocks = context.hidden_layer.size();
		num_threads = sample.size();
		cuda_neural_network<<<num_blocks, num_threads, num_threads * sizeof(float)>>>((&context.dev_input) + i * sample.size(), &context.dev_hidden, &context.dev_weights);
		hipDeviceSynchronize();

		if (hipSuccess != hipGetLastError()) {
			return false;
		}

		num_blocks = context.output_layer.size();
		num_threads = context.hidden_layer.size();
		cuda_neural_network<<<num_blocks, num_threads, num_threads * sizeof(float)>>>(&context.dev_hidden, &context.dev_output, (&context.dev_weights) + ((sample.size() + 1) * context.hidden_layer.size()));
		hipDeviceSynchronize();

		if (hipSuccess != hipGetLastError()) {
			return false;
		}

		num_blocks = 1;
		num_threads = context.output_layer.size();
		cuda_neural_network_output_error<<<num_blocks, num_threads>>>(&context.dev_output, (&context.dev_labels) + i * context.output_layer.size());
		hipDeviceSynchronize();

		if (hipSuccess != hipGetLastError()) {
			return false;
		}

		num_blocks = context.hidden_layer.size();
		num_threads = context.output_layer.size();
		context.dev_mode.synchToDevice(1);
		cuda_neural_network_error<<<num_blocks, num_threads, num_threads * sizeof(float)>>>(&context.dev_hidden, &context.dev_output, (&context.dev_weights) + ((sample.size() + 1) * context.hidden_layer.size()), &context.dev_learning, &context.dev_mode);
		hipDeviceSynchronize();

		if (hipSuccess != hipGetLastError()) {
			return false;
		}

		num_blocks = sample.size();
		num_threads = context.hidden_layer.size();
		context.dev_mode.synchToDevice(0);
		cuda_neural_network_error<<<num_blocks, num_threads, num_threads * sizeof(float)>>>((&context.dev_input) + i * sample.size(), &context.dev_hidden, &context.dev_weights, &context.dev_learning, &context.dev_mode);
		hipDeviceSynchronize();

		if (hipSuccess != hipGetLastError()) {
			return false;
		}

		return true;
	}

	bool neuronalNetwork::test_sample(const int i, const data::sample<float>& sample, data_context& context, cuda::model& model) {
		int num_blocks;
		int num_threads;

		num_blocks = context.hidden_layer.size();
		num_threads = sample.size();
		cuda_neural_network<<<num_blocks, num_threads, num_threads * sizeof(float)>>>((&context.dev_input) + i * sample.size(), &context.dev_hidden, &context.dev_weights);
		hipDeviceSynchronize();

		if (hipSuccess != hipGetLastError()) {
			return false;
		}

		num_blocks = context.output_layer.size();
		num_threads = context.hidden_layer.size();
		cuda_neural_network<<<num_blocks, num_threads, num_threads * sizeof(float)>>>(&context.dev_hidden, &context.dev_output, (&context.dev_weights) + ((sample.size() + 1) * context.hidden_layer.size()));
		hipDeviceSynchronize();

		if (hipSuccess != hipGetLastError()) {
			return false;
		}

		context.dev_output.synchFromDevice(context.output_layer);
		hipDeviceSynchronize();

		int maxIndex = std::distance(context.output_layer.begin(), std::max_element(context.output_layer.begin(), context.output_layer.end()));

		return (sample.getLabel() == maxIndex);
	}


	neuronalNetwork::data_context::data_context(const config_t config, const cuda::model& model, const std::vector<data::sample<float>>& samples)
	: hidden_layer(config.num_hidden),
	output_layer(config.num_output),
	labels(samples.size() * output_layer.size()),
	dev_input(samples[0].internalData(), samples.size()),
	dev_hidden(hidden_layer),
	dev_output(output_layer),
	dev_weights(model.getWeights()),
	dev_labels(labels)
	{
		for (int i = 0; i < samples.size(); i++) {
			for (int j = 0; j < output_layer.size(); j++) {
				if (samples[i].getLabel() == j) {
					labels[i * output_layer.size() + j] = 1;
				} else {
					labels[i * output_layer.size() + j] = 0;
				}
			}
		}
	}

	void neuronalNetwork::data_context::synchronize(const config_t config, const cuda::model& model, const std::vector<data::sample<float>>& samples) {
		for (int i = 0; i < samples.size(); i++) {
			dev_input.synchToDevice(samples[i].internalData(), i);
		}

		dev_hidden.synchToDevice(hidden_layer);
		dev_output.synchToDevice(output_layer);
		dev_weights.synchToDevice(model.getWeights());
		dev_labels.synchToDevice(labels);
		dev_learning.synchToDevice(config.learning_rate);
	}

}
