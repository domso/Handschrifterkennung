#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include "NeuronalNetwork.h"
#include "sample_set.h"
#include "sample.h"


/**
 * Training the network by processing the MNIST training set and updating the weights
 * @param nn
 */
void trainNetwork(NeuronalNetwork* nn, std::vector<data::sample<float>> trainingSamples){
	int errorCount = 0;

	for(int i = 0; i < trainingSamples.size(); i++){
		std::vector<float>& input = trainingSamples[i].internalData();
		int label = trainingSamples[i].getLabel();

		nn->feedInput(input);

		nn->feedForwardNetwork();

		nn->backPropagateNetwork(label);

		int classification = nn->getNetworkClassification();
		if(classification != label){
			std::cout << "network computed " << classification << ", but label is " << label << "\n";
			errorCount++;
		}
	}
	std::cout << "training completed!\n => " << errorCount << " mistakes out of " << trainingSamples.size() << " images\n";
}

int main(int argc, char** argv) {
	std::vector<data::sample<float>> trainingInput = data::sample_set::load<float>("./train-images.idx3-ubyte", "./train-labels.idx1-ubyte");

	int imgCount    = trainingInput.size();
	if(imgCount == 0){
		std::cout << "no images were loaded, exiting....";
		return -1;
	}

	int inputCount  = trainingInput[0].size();
	int hiddenCount = 20;
	int outputCount = 10;

	NeuronalNetwork* nn = new NeuronalNetwork(inputCount, hiddenCount, outputCount);

	for(int i = 0; i < 10; i++)
		trainNetwork(nn, trainingInput);

	delete nn;
}
